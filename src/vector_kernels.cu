#include "hip/hip_runtime.h"

#include <vector_kernels.h>
#include<iostream>
#include<math.h>


#define vector_size 400000
#define numThreads 1024 // 500 threads in a block
//#define numBlocks 2  // 2 blocks

__global__
void vector_add_kernel(int * a, int * b ,int * result)

{
    uint32_t ind = blockDim.x * blockIdx.x + threadIdx.x;

    if(ind < vector_size)
        result[ind] = a[ind] * b[ind];
}



void run_vector_add_kernel()
{
    // allocate memory on the cpu
    int * h_a = (int *) malloc(sizeof(int)*vector_size);
    int * h_b = (int *) malloc(sizeof(int)*vector_size);
    int * h_c = (int *) malloc(sizeof(int)*vector_size);

    // init vector a & b , when add them the answer would be 1 in alll cells

    for(int i = 0 ; i<vector_size ;i++ )
        {
           
                h_a[i] = 1;
                h_b[i] = 1;
           
            
        }

       


    // allocate memory on the gpu
    int * d_a ;
    int * d_b ;
    int * d_c ;
    hipMalloc(&d_a,sizeof(int)*vector_size);
    hipMalloc(&d_b,sizeof(int)*vector_size);
    hipMalloc(&d_c,sizeof(int)*vector_size);


    // copy host vectors to device vectors
    hipMemcpy( d_a, h_a, vector_size * sizeof(int),hipMemcpyHostToDevice );
    hipMemcpy( d_b, h_b, vector_size * sizeof(int),hipMemcpyHostToDevice );

    // run the kernel

    dim3 grid_size = (int)ceil(vector_size / numThreads);
    dim3 block_size = numThreads;

    vector_add_kernel<<<grid_size,block_size>>>(d_a , d_b , d_c);


    // copy the result from device to host
    hipMemcpy( h_c , d_c, vector_size * sizeof(int),hipMemcpyDeviceToHost );

    //check the answer
    for (int i = 0 ; i < vector_size ; i++)
        {
            std::cout<<h_c[i] << ' ';
        }
        std::cout<<std::endl;


    // free memory cpu
    delete  h_a;
    delete [] h_b;
    delete [] h_c;

    //free memory gpu
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);



}



