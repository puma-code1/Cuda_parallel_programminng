#include "hip/hip_runtime.h"

#include<matrix_kernels.h>
#include<iostream>
// matrix muptipication

//dim3 blockDim(size_t blockDimX, size_t blockDimY) 16X16  - 300 blocks of 256  
//dim3 gridDim(size_t gridDimX, size_t gridDimY)  30X30
// or we can do 32X32 blockDim, 15X15 gridDim  - 225 blocks of 1024
// 



__global__ void mm_kernel(float* A, float* B, float* C, int n)
{
    float temp_sum = 0 ;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < n && col < n) {
        for (int i = 0; i < n; ++i) {
             temp_sum += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = temp_sum;
    }
}
//mm_kernel << <dimGrid, dimBlock >> > (d_a, d_b, d_c, n);



void run_mm_kernel()
{
    
}


// if below threashold make it zero
__global__ 
void travese_threshold_matrix_kernel(uint16_t * mat, int threshold, uint32_t size )
{

   
   // very important traverse each pixel with mul of y
    int index = blockDim.x * blockIdx.x + threadIdx.x + 480*(blockDim.y * blockIdx.y + threadIdx.y);
    //int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < size && mat[index ]< threshold)
        {
         mat[index] = 33;
        }
   
}

void run_travese_threshold_matrix_kernel(uint16_t * mat, int threshold, uint32_t mat_size)
{

    // allocate memory on the cpu
    uint16_t * h_a = (uint16_t *) malloc(sizeof(uint16_t)*mat_size);
    

    // init mat test

    for(int i = 0 ; i<mat_size ;i++ )
        {
           if(i & 1)
                h_a[i] = 100;
            else
                h_a[i] = 10;
            
        }

    // allocate memory on the gpu
    uint16_t * d_a ;
    
    hipMalloc(&d_a,sizeof(uint16_t)*mat_size);
    
    // copy host vectors to device vectors
    hipMemcpy( d_a, h_a, mat_size * sizeof(uint16_t),hipMemcpyHostToDevice );


    // run the kernel
     dim3 grid_size_ (16,16); //256; //  ;//grid_size_ = 256;  // 256 blocks
     dim3 block_size_ (32,32); // 1024 threads in block
    travese_threshold_matrix_kernel <<<grid_size_,block_size_>>>(d_a, threshold, mat_size);

    // copy the result from device to host
    hipError_t err  = hipMemcpy( h_a , d_a, mat_size * sizeof(uint16_t),hipMemcpyDeviceToHost );

    //check the answer
    for (int i = 0 ; i < mat_size ; i++)
        {
            
                std::cout<<h_a[i] <<" ";
            
            
        }
        std::cout<<std::endl;


    // free memory cpu
    delete  h_a;

    //free memory gpu
    hipFree(d_a);
  
}